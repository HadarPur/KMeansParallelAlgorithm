#include "hip/hip_runtime.h"
#include "KMeans Header.h"
#include <stdio.h>

__global__ void callPointsCoordinatesByTimeWithCuda(Point* points, int numOfPoints, double time)
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;

	int threadIndex = bid*CUDA_BLOCK_SIZE + tid;

	if (threadIndex < numOfPoints)
	{
		points[threadIndex].x = points[threadIndex].x0 + points[threadIndex].vx * time;
		points[threadIndex].y = points[threadIndex].y0 + points[threadIndex].vy * time;
		points[threadIndex].z = points[threadIndex].z0 + points[threadIndex].vz * time;
	}
}

void callPointsCoordinatesWithCuda(Point* points, int numOfPoints, double time)
{
	int numOfBlocks;
	Point* device_points;
	hipError_t cudaStatus;

	//numOfBlocks = get_block_num(numOfPoints);
	numOfBlocks = numOfPoints / CUDA_BLOCK_SIZE;
	if (numOfPoints % CUDA_BLOCK_SIZE > 0)
		numOfBlocks++;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		hipFree(device_points);
	}

	// Allocate memory on GPU 
	cudaStatus = hipMalloc((void**)&device_points, numOfPoints * sizeof(Point));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dpPoints failed!");
		hipFree(device_points);
	}

	// Copy memory from CPU to GPU
	cudaStatus = hipMemcpy(device_points, points, numOfPoints * sizeof(Point), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy dpPoints failed!");
		hipFree(device_points);
	}

	// kernel function,each thread gets: dpPoints, part_size, dT
	callPointsCoordinatesByTimeWithCuda << <numOfBlocks, CUDA_BLOCK_SIZE >> >(device_points, numOfPoints, time);

	// Check errors
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "perform_deltaT_movements launch failed: %s\n", hipGetErrorString(cudaStatus));
		hipFree(device_points);
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns errors
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching perform_deltaT_movements!\n", cudaStatus);
		hipFree(device_points);
	}

	// Copy memory from GPU to CPU memory
	cudaStatus = hipMemcpy(points, device_points, numOfPoints * sizeof(Point), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy from device  failed!");
		hipFree(device_points);
	}
	hipFree(device_points);
}